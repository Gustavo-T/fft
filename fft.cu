#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include "utl.cuh"

#include <math.h>
#include <cstring>

#include <windows.h>

#define SIZE 1024*1024*2
#define THREADS_PER_BLOCK 256
#define BLOCKS 32


//-----------------Complex multiply------------------
static __device__ __host__ inline hipfftComplex complexMul(hipfftComplex a, hipfftComplex b)
{
	float tmp = 0.0f;
	tmp = a.x * b.x - a.y*(-1)*b.y;
	a.y = a.x *(-1)* b.y + b.x * a.y;
	a.x = tmp;

	return a;
}
//-------------------------------------
static __global__ void multiply(hipfftComplex* a, hipfftComplex* b, int size)
{

	const int numThreads = blockDim.x * gridDim.x;
	const int threadId = blockIdx.x * blockDim.x + threadIdx.x;

	for (size_t i = threadId;  i < size; i+= numThreads ) {
		a[i] = complexMul(a[i],b[i]);

	}
}
//-------------------------------------

bool memcpCheck(hipfftComplex* a, hipfftComplex* d_a, size_t size ) 
{
	checkCudaErrors(hipMemcpy(d_a, a,sizeof(hipfftComplex) * size, hipMemcpyHostToDevice));
	hipfftComplex* tData = new hipfftComplex[size];
	checkCudaErrors(hipMemcpy(tData, d_a,sizeof(hipfftComplex) * size, hipMemcpyDeviceToHost));
	return  (memcmp(a,tData,sizeof(hipfftComplex) * size) == 0) ? true : false;

}

int main()
{
	int dev = 0;
	printf("Input device number ");
	scanf("%d", &dev);

	hipSetDevice(dev);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,dev);
	printf("\nName %s\n", deviceProp.name);


	//////////////

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	float timeStartMem, timeMem = 0; 
	
	int logSize = log(2.0*SIZE)/log(2.0) + 1;
	size_t TSIZE = pow(2.0, logSize);
	size_t maxLag = SIZE - 1;


	hipfftComplex  *IndataX = new hipfftComplex[SIZE];
	hipfftComplex  *IndataY = new hipfftComplex[SIZE];

	hipfftComplex *dataX = new hipfftComplex[TSIZE];
	hipfftComplex *dataY = new hipfftComplex[TSIZE];

	hipfftComplex *resultX = new hipfftComplex[TSIZE];
	hipfftComplex *result = new hipfftComplex[TSIZE];

	hipfftHandle forwardPlanX, forwardPlanY, backwardPlan;
	
	// create a simple plan for 1D transform
	if(hipfftPlan1d(&forwardPlanX,TSIZE,HIPFFT_C2C,1) != HIPFFT_SUCCESS) { printf("Cuda: hipfftPlan1d HIPFFT_C2C failed\n"); return 1; }
	if(hipfftPlan1d(&forwardPlanY,TSIZE,HIPFFT_C2C,1) != HIPFFT_SUCCESS) { printf("Cuda: hipfftPlan1d HIPFFT_C2C failed\n"); return 1; }
	if (hipfftPlan1d(&backwardPlan,TSIZE,HIPFFT_C2C,1) != HIPFFT_SUCCESS) { printf("Cuda: hipfftPlan1d HIPFFT_C2C failed\n"); return 1; }
	///
	// file open
	FILE* inFileX = fopen("file1.sig","rb");

	if(!inFileX)
	{
		printf("Can't open file %s\n","file1.sig");
		exit(0);
	}


	FILE* inFileY = fopen("file2.sig","rb");

	if(!inFileY)
	{
		printf("Can't open file %s\n","file2.sig");
		exit(0);
	}

		short int xreal = 0,yreal = 0;
		short int xim = 0,yim = 0;

	//download data from files
	for(size_t i = 0; i < SIZE; ++i)
	{
		
		fread(&xreal, sizeof( unsigned short ), 1, inFileX);
		fread(&yreal, sizeof(unsigned short), 1, inFileY);

		fread(&xim, sizeof( unsigned short ), 1, inFileX);
		fread(&yim, sizeof( unsigned short ), 1, inFileY);

		IndataX[i].x = (float) xreal;
		IndataX[i].y = (float) xim;

		IndataY[i].x = (float) yreal;
		IndataY[i].y = (float) yim;

	}

	fclose(inFileX);
	fclose(inFileY);

	hipfftComplex XMean,YMean;
	memset(&XMean,0,sizeof(hipfftComplex));
	memset(&YMean,0,sizeof(hipfftComplex));

	for(size_t i = 1; i < SIZE; ++i)
	{

		dataX[i-1].x = IndataX[i].x * IndataX[i-1].x - IndataX[i].y * (-1)* IndataX[i-1].y; 
		dataX[i-1].y = IndataX[i].x * (-1) * IndataX[i-1].y + IndataX[i-1].x * IndataX[i].y;

		dataY[i-1].x = IndataY[i].x * IndataY[i-1].x - IndataY[i].y * (-1) * IndataY[i-1].y;
		dataY[i-1].y = IndataY[i].x * (-1) * IndataY[i-1].y + IndataY[i-1].x * IndataY[i].y;

		XMean.x += dataX[i-1].x; 
		XMean.y += dataX[i-1].y;

		YMean.x += dataY[i-1].x; 
		YMean.y += dataY[i-1].y;

	}

	XMean.x = XMean.x/(SIZE-1);
	XMean.y = XMean.y/(SIZE-1);

	YMean.x = YMean.x/(SIZE-1);
	YMean.y = YMean.y/(SIZE-1);


	for(size_t i = 0; i < maxLag; ++i)
	{
		dataX[i].x-= XMean.x; 
		dataX[i].y-= XMean.y;

		dataY[i].x-= YMean.x; 
		dataY[i].y-= YMean.y;
	}

	float SigmaX = 0.0f;
	float SigmaY = 0.0f;

	//compute sigma
	for(size_t i = 0; i < maxLag; ++i)
	{
		SigmaX += (dataX[i].x*dataX[i].x) + (dataX[i].y*dataX[i].y);
		SigmaY += (dataY[i].x*dataY[i].x) + (dataY[i].y*dataY[i].y);
	}

	SigmaX = sqrt(SigmaX);
	SigmaY = sqrt(SigmaY);


	//pad vector with 0's
	for(size_t i = maxLag; i < TSIZE; ++i)
	{
		dataX[i].x = 0;
		dataX[i].y = 0;

		dataY[i].x = 0; 
		dataY[i].y = 0;

	}

	hipfftComplex *d_dataX, *d_dataY;

	long sizeData = 0;
	checkCudaErrors(hipMalloc(&d_dataX, sizeof(hipfftComplex) * TSIZE));
	
	hipEventRecord(start, 0);
	// Copy host memory to device
	checkCudaErrors(hipMemcpy(d_dataX, dataX,sizeof(hipfftComplex) * TSIZE, hipMemcpyHostToDevice));
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&timeStartMem, start, stop);
	timeMem = timeStartMem;
	
	sizeData = sizeof(hipfftComplex) * TSIZE;


	checkCudaErrors(hipMalloc(&d_dataY, sizeof(hipfftComplex) * TSIZE));

	if(memcpCheck(dataX,d_dataX,TSIZE)) 
	{
		hipEventRecord(start, 0);
		checkCudaErrors(hipMemcpy(d_dataY, dataY,sizeof(hipfftComplex) * TSIZE, hipMemcpyHostToDevice));
		hipEventRecord(stop, 0); 
		hipEventSynchronize(stop); 
		hipEventElapsedTime(&timeStartMem, start, stop);
		timeMem += timeStartMem;
		
	}
	
	float elapsedTime, execTime = 0; // time execute cufftExec

	hipEventRecord(start, 0);
	// execute plan for forward fft
	if ( hipfftExecC2C(forwardPlanX,d_dataX,d_dataX,HIPFFT_FORWARD) != HIPFFT_SUCCESS) { printf("Cuda: hipfftExecC2C failed\n"); return 1; }
	
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	
	hipEventElapsedTime(&elapsedTime, start, stop);
	execTime = elapsedTime;
	
	

	//////////////////////////////////////////////////////

	hipEventRecord(start, 0);

	checkCudaErrors(hipMemcpy(dataX, d_dataX,sizeof(hipfftComplex) * TSIZE, hipMemcpyDeviceToHost));

	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&timeStartMem, start, stop);
	timeMem += timeStartMem;

	hipEventRecord(start, 0);
	// execute plan for forward fft
	if ( hipfftExecC2C(forwardPlanY,d_dataY,d_dataY,HIPFFT_FORWARD) != HIPFFT_SUCCESS) { printf("Cuda: hipfftExecC2C failed\n"); return 1; }
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);

	hipEventElapsedTime(&elapsedTime, start, stop);
	execTime += elapsedTime;
	
	hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(dataY, d_dataY,sizeof(hipfftComplex) * TSIZE, hipMemcpyDeviceToHost));
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&timeStartMem, start, stop);
	timeMem += timeStartMem;
	
	hipfftComplex *d_result;
	
	checkCudaErrors(hipMalloc(&d_result, sizeof(hipfftComplex) * TSIZE));

	hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(d_result, dataX,sizeof(hipfftComplex) * TSIZE, hipMemcpyHostToDevice));
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&timeStartMem, start, stop);
	timeMem += timeStartMem;
	
	printf("sizeof(dataX)= %d bytes\n", TSIZE);
	
	multiply<<<BLOCKS,THREADS_PER_BLOCK>>>(d_dataX,d_dataY,TSIZE);

	hipEventRecord(start, 0);
	// execute plan for inverse fft
	if ( hipfftExecC2C(backwardPlan,d_dataX,d_result,HIPFFT_BACKWARD) != HIPFFT_SUCCESS) { printf("Cuda: hipfftExecC2C failed\n"); return 1; }
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	execTime += elapsedTime;

	hipDeviceSynchronize();

	hipEventRecord(start, 0);
	checkCudaErrors(hipMemcpy(result,d_result,sizeof(hipfftComplex) * TSIZE,hipMemcpyDeviceToHost));
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&timeStartMem, start, stop);
	timeMem += timeStartMem;
	
	for(size_t i = 0; i < TSIZE; ++i)
	{
		result[i].x = result[i].x/SigmaX;
		result[i].x = result[i].x/SigmaY;
		result[i].x = result[i].x/TSIZE;

		result[i].y = result[i].y/SigmaX;
		result[i].y = result[i].y/SigmaY;
		result[i].y = result[i].y/TSIZE;

	}

	hipfftComplex *Result =  new hipfftComplex[TSIZE];

	//swap positive and negative lags
	for(unsigned int i = 0; i < maxLag; ++i)
	{
		memcpy(&Result[i].x,&result[TSIZE - maxLag + 1 + i].x,sizeof(hipfftComplex) );
		memcpy(&Result[i].y,&result[TSIZE - maxLag + 1 + i].y,sizeof(hipfftComplex) );
	}

	for(size_t i = 0; i < maxLag; ++i)
	{
		Result[i + maxLag].x = result[i].x; 
		Result[i + maxLag].y = result[i].y;
	}

	float max = 0;
	size_t maxindex;

	for(size_t i = 0; i < 2*maxLag; ++i)
	{
		float temp = sqrt(Result[i].x*Result[i].x + Result[i].y*Result[i].y);
		
		if (temp > max)
		{
			max = temp;
			maxindex = i;
		}
		
	}

	double res = 3*sizeData/(timeMem*0.001);
	long r = res;
	printf("Max element %1.7f, index %d", max , maxindex+1);

	// free
	hipFree(d_dataX);
	hipFree(d_dataY);
	hipFree(d_result);

	delete []IndataX;
	delete []IndataY;
	delete []dataX;
	delete []dataY;
	delete []resultX;
	delete []result;
	delete []Result;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipfftDestroy(forwardPlanX);
	hipfftDestroy(forwardPlanY);
	hipfftDestroy(backwardPlan);

	// print results
	int te = execTime, tm = timeMem;
	
	printf("\nTime mem, ms %d\n", tm);
	printf("Time exec,ms = %d\n", te);
	printf("Velocity, mb/s %ul \n", r/(1024*1024));
	///

	system("pause"); // if necessary
	return 0;
}

